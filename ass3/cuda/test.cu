#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <limits.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "hostlib.cu.h"

// in general
#define EPS 0.0005
#define BLOCK_SIZE 512
// for warmup
#define NUM_THREADS 7624
// task one specific
#define COLS_N 32
#define ROWS_M 32
// task two specific
#define N2 32
#define M2 64 // As per assignment
// task three specific
#define N 32 
#define M 32
#define U 32


// declaration of functions used in main
// ALL should be moved to hostlib and implemented there
bool validate(float* ground_truth, float* same);    // TODO

int transpose_cpu(int rows_in, int cols_in, float *m_in, float *m_out);       // TODO
int transpose_gpu_naive(int rows_in, int cols_in, float *m_in, float *m_out); // TODO
int transpose_gpu(int rows_in, int cols_in, float *m_in, float *m_out);       // TODO

int matrix_accfun_cpu(int rows_in, int cols_in, float* m_in, float* m_out_a);        // TODO 
int matrix_accfun_gpu_first(int rows_in, int cols_in, float* m_in, float* m_out_a);  // TODO
int matrix_accfun_gpu_second(int rows_in, int cols_in, float* m_in, float* m_out_a); // TODO

int matmult_cpu(int M, int U, float* m_in_a, int U, int N, float* m_in_b, float* m_out_a);     // TODO
int matmult_gpu(int M, int U, float* m_in_a, int U, int N, float* m_in_b, float* m_out_a);     // TODO
int matmult_gpu_opt(int M, int U, float* m_in_a, int U, int N, float* m_in_b, float* m_out_a); // TODO



// declared with purpose of starting the file with its main function
int warmup();
int task_one();
int task_two();
int task_three();


int main(int argc, char** argv) {
    warmup(); // sole purpose is 'warming up GPU' so that timings get valid downstream.
    task_one();
    task_two();
    task_three();
}




int warmup(){
    // performing max segment sum calculation for GPU warmup purpose
    const unsigned int block_size  = BLOCK_SIZE;
    int* h_in    = (int*) malloc( NUM_THREADS * sizeof(int));

    for(unsigned int i=0; i<NUM_THREADS; i++) h_in[i] = 1;

    { // calling maxSegmentSum
        int* d_in;
        hipMalloc((void**)&d_in , NUM_THREADS * sizeof(int));

        // copy host memory to device
        hipMemcpy(d_in, h_in, mem_size_int, hipMemcpyHostToDevice);

        // run function on GPU
        maxSegmentSum ( block_size, num_threads, d_in );
        
        // cleanup memory
        hipFree(d_in );
    }

    // cleanup memory
    free(h_in );

    return 0;
}



int task_one(){
    // Transpose Matrix 
    // 1a. implement serial version
    // 1b. bonus objective implement in OPENMP
    // 1c. implement naive  version
    // 1d. implement serial version

    // initiate data to transpose (dense matrix)
    float *h_in, *h_out_a, *h_out_b, *h_out_c, *h_out_d;
    m_in = malloc(COLS_N * ROWS_M * sizeof(float));
    m_out_a = malloc(COLS_N * ROWS_M * sizeof(float));
    m_out_c = malloc(COLS_N * ROWS_M * sizeof(float));
    m_out_d = malloc(COLS_N * ROWS_M * sizeof(float));

    for (int i=0; i<(COLS_N*ROWS_M); i++){
        m_in[i] = 0 // TODO random number
    }

    // initiate timing variable, keep results for validation
    unsigned long int elapsed_a, elapsed_b, elapsed_c, elapsed_d;
    struct timeval t_start, t_end, t_diff;
    bool valid_c, valid_d; 
    
    // TASK 1 A)
    { 
        gettimeofday(&t_start, NULL); 

        transpose_cpu(ROWS_M, COLS_N, m_in, m_out_a);
    
        gettimeofday(&t_end, NULL); 
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_a = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
    }
    printf("Transpose Matrix sized %d x %d on CPU runs in: %lu microsecs", COLS_N, ROWS_M, elapsed_a);
    
    // TASK 1 B) OMITTED

    // TASK 1 C)
    { 
        gettimeofday(&t_start, NULL); 

        transpose_gpu_naive(ROWS_M, COLS_N, m_in, m_out_a);
    
        gettimeofday(&t_end, NULL); 
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_c = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
	valid_c = validate(h_out_a, h_out_c);
    }
    printf("Transpose Matrix sized %d x %d on GPU naïvely runs in: %lu microsecs\n", COLS_N, ROWS_M, elapsed_c);
    if (valid_c) printf("Naïve implementation is VALID\n");
    else printf("Naïve implementation is INVALID\n");


    // TASK 1 D)
    { 
        gettimeofday(&t_start, NULL); 

        transpose_gpu(ROWS_M, COLS_N, m_in, m_out_a);
    
        gettimeofday(&t_end, NULL); 
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_d = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
	valid_d = validate(h_out_a, h_out_d);
    }
    printf("Transpose Matrix sized %d x %d on GPU optimized runs in: %lu microsecs", COLS_N, ROWS_M, elapsed_d);
    if (valid_d) printf("Optimal implementation is VALID\n");
    else printf("Optimal implementation is INVALID\n");

    // TODO print statistics, speedup difference and so on

    free(h_in);
    free(h_out_a);
    free(h_out_c);
    free(h_out_d);

    return 0;
}




int task_two(){
    // Matrix Transposition during or as a pre-computatiion
    // 2 a. Reason about loop-level parallellism 
    // 2 b. bonus objective implement in OPENMP
    // 2 c. implement QUICKLY straightforward cuda
    // 2 d. Rewrite QUICKLY to coalesced global mem access

    // initiate data to transpose (dense matrix)
    const int m = 64;
    float *h_in, *h_out_a, *h_out_b, *h_out_c, *h_out_d;
    m_in = malloc(M2 * N2 * sizeof(float));
    m_out_a = malloc(M2 * N2 * sizeof(float));
    m_out_c = malloc(M2 * N2 * sizeof(float));
    m_out_d = malloc(M2 * N2 * sizeof(float));

    for (int i=0; i<(M2*N2); i++){
        m_in[i] = 0 // TODO random number
    }

    // initiate timing variable, keep results for validation
    unsigned long int elapsed_a, elapsed_b, elapsed_c, elapsed_d;
    struct timeval t_start, t_end, t_diff;
    bool valid_c, valid_d; 
    
    // TASK 1 A)
    { 
        gettimeofday(&t_start, NULL); 

        matrix_accfun_cpu(M2, N2, m_in, m_out_a);
    
        gettimeofday(&t_end, NULL); 
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_a = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
    }
    printf("Matrix accfun on size %d x %d on CPU runs in: %lu microsecs",M2, N2, elapsed_a);
    
    // TASK 1 B) OMITTED

    // TASK 1 C)
    { 
        gettimeofday(&t_start, NULL); 

        matrix_accfun_gpu_first(M2, N2, m_in, m_out_c); 
    
        gettimeofday(&t_end, NULL); 
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_c = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
	valid_c = validate(h_out_a, h_out_c);
    }
    printf("Matrix accfun on size %d x %d on GPU first impl runs in: %lu microsecs\n",M2, N2, elapsed_c);
    if (valid_c) printf("Implementation is VALID\n");
    else printf("Implementation is INVALID\n");


    // TASK 1 D)
    { 
        gettimeofday(&t_start, NULL); 

        matrix_accfun_gpu_first(M2, N2, m_in, m_out_d);  
    
        gettimeofday(&t_end, NULL); 
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_d = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
	valid_d = validate(h_out_a, h_out_d);
    }
    printf("Matrix accfun on size %d x %d on GPU rewrite runs in: %lu microsecs",M2, N2, elapsed_d);
    if (valid_d) printf("Implementation is VALID\n");
    else printf("Implementation is INVALID\n");

    // TODO print statistics, speedup difference and so on

    free(h_in);
    free(h_out_a);
    free(h_out_c);
    free(h_out_d);

    return 0;
}



int task_three(){
    // Dense Matrix-Matrix multiplication
    // 1a. implement serial version
    // 1b. bonus objective implement in OPENMP
    // 1c. implement naive  version
    // 1d. implement serial version

    // initiate data to transpose (dense matrix)
    float *m_in_a, *m_in_b, *m_out_a, *m_out_c, *m_out_d;
    m_in_a = malloc(M * U * sizeof(float));
    m_in_b = malloc(U * N * sizeof(float));
    m_out_a = malloc(M * N * sizeof(float));
    m_out_c = malloc(M * N * sizeof(float));
    m_out_d = malloc(M * N * sizeof(float));
    
    for (int i=0; i<(COLS_N*ROWS_M); i++){
        m_in[i] = 0; // TODO random number
    }

    // initiate timing variable, keep results for validation
    unsigned long int elapsed_a, elapsed_b, elapsed_c, elapsed_d;
    struct timeval t_start, t_end, t_diff;
    bool valid_c, valid_d; 
    
    // TASK 1 A)
    { 
        gettimeofday(&t_start, NULL); 

        matmult_cpu(M, U, m_in_a, U, N, m_in_b, m_out_a);
    
        gettimeofday(&t_end, NULL); 
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_a = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
    }
    printf("MatrixMult on (%dx%d) x (%d,%d) on CPU runs in: %lu microsecs",M,U,U,N, elapsed_a);
    
    // TASK 1 B) OMITTED

    // TASK 1 C)
    { 
        gettimeofday(&t_start, NULL); 

        matmult_gpu(M, U, m_in_a, U, N, m_in_b, m_out_a);
    
        gettimeofday(&t_end, NULL); 
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_c = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
	valid_c = validate(h_out_a, h_out_c);
    }
    printf("MatrixMult on (%dx%d) x (%d,%d) on GPU runs in: %lu microsecs",M,U,U,N, elapsed_c);
    if (valid_c) printf("Implementation is VALID\n");
    else printf("Implementation is INVALID\n");


    // TASK 1 D)
    { 
        gettimeofday(&t_start, NULL); 

        matmult_gpu_opt(M, U, m_in_a, U, N, m_in_b, m_out_a);
    
        gettimeofday(&t_end, NULL); 
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed_d = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
	valid_d = validate(h_out_a, h_out_d);
    }
    printf("MatrixMult on (%dx%d) x (%d,%d) on CPU runs in: %lu microsecs",M,U,U,N, elapsed_d);
    if (valid_d) printf("Optimal Implementation is VALID\n");
    else printf("Optimal Implementation is INVALID\n");

    // TODO print statistics, speedup difference and so on

    free(h_in);
    free(h_out_a);
    free(h_out_c);
    free(h_out_d);

    return 0;
}

