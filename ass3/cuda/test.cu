#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <limits.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "hostlib.cu.h"




// declared with purpose of starting the file with its main function
void task_one(int M, int N, bool verbose);
void task_two(int M, int N, bool verbose);
void task_three(int M, int U, int N, bool verbose);
void warmup(const unsigned int size);


// main test program
int main(int argc, char** argv) {
    warmup(1024); // sole purpose is 'warming up GPU' so that timings get valid downstream.
    task_one(   (8*1024)   , (8*1024), false);
    task_two(   (6*64*1024), (64)    , false);
    task_three( (5*256)    , (5*256) , (5*256), false);
    return 0;
}






void task_one(int M, int N, bool verbose){
    printf("\n\nASSIGNMENT3 TASK1: MATRIX TRANSPOSITION\n");
    // Transpose Matrix 
    // 1a. implement serial version
    // 1b. bonus objective implement in OPENMP
    // 1c. implement naive  version
    // 1d. implement serial version

    // initiate data to transpose (dense matrix)
    const unsigned int rows = M;
    const unsigned int cols = N;
    const char s_valid[10]   = "--  VALID";
    const char s_invalid[10] = "--INVALID";
    const unsigned int arr_size = rows * cols;

    float *m_in, *m_out_a, *m_out_c, *m_out_d;
    m_in    = (float*) malloc(arr_size * sizeof(float));
    m_out_a = (float*) malloc(arr_size * sizeof(float));
    m_out_c = (float*) malloc(arr_size * sizeof(float));
    m_out_d = (float*) malloc(arr_size * sizeof(float));

    // INITIALIZE MATRIX (ARRAY) TO WORK ON
    for (int i=0; i<(arr_size); i++){
        m_in[i] = (float) i;
    }

    unsigned long int elapsed_a, elapsed_c, elapsed_d;
    bool valid_c, valid_d; 
    
    // TASK 1 A)
    {
        elapsed_a = transpose_cpu<float>( rows, cols, m_in, m_out_a);
    }
    
    // TASK 1 B) OMITTED

    // TASK 1 C)
    { 
        const unsigned char naive_version = 1;
        elapsed_c = transpose_gpu<float>( rows, cols, m_in, m_out_c, naive_version);
	valid_c   = mvalidate<float>(rows, cols, m_out_a, m_out_c);
    }

    // TASK 1 D)
    { 
        const unsigned char opt_version = 2;
        elapsed_d = transpose_gpu<float>( rows, cols, m_in, m_out_d, opt_version);
	valid_d   = mvalidate<float>( rows, cols, m_out_a, m_out_d);
    }

    // print statistics
    const double gpu_speedup  = ((double) elapsed_c) / ((double) elapsed_d + 1);
    const double cpu_speedup1 = ((double) elapsed_a) / ((double) elapsed_c + 1);
    const double cpu_speedup2 = ((double) elapsed_a) / ((double) elapsed_d + 1);
    const unsigned int mops = 2 * rows * cols;
    float gmops_a = (float) mops / ( elapsed_a * 1000);
    float gmops_c = (float) mops / ( elapsed_c * 1000);
    float gmops_d = (float) mops / ( elapsed_d * 1000);

    printf("\nTranspose Matrix sized %d x %d running times\n", cols, rows);
    printf("CPU:           %10lu microsecs. \n", elapsed_a);
    printf("GPU naïve:     %10lu microsecs. %s\n", elapsed_c, (valid_c ? s_valid: s_invalid));
    printf("GPU optimized: %10lu microsecs. %s\n", elapsed_d, (valid_d ? s_valid: s_invalid));

    printf("\nGiga MemoryOPerations per second:\n");
    printf("CPU:           %10.3f Gmop/s.\n", gmops_a);
    printf("GPU naïve:     %10.3f Gmop/s.\n", gmops_c);
    printf("GPU optimized: %10.3f Gmop/s.\n", gmops_d);

    printf("\nThis is a speedup of %7.2f, for tile and coalesced mem accesses on GPU.\n",gpu_speedup);
    printf("... and a speedup of %7.2f, for GPU naive compared to CPU.\n",cpu_speedup1);
    printf("... and a speedup of %7.2f, for GPU opt   compared to CPU.\n",cpu_speedup2);

    if (verbose) { // also print output matrices
        printf("\nInput matrix before transposition: \n");
        matprint(rows, cols, m_in);
        printf("Matrix after transposition by cpu: \n");
        matprint(cols, rows, m_out_a);
        printf("Matrix after transposition by gpu (naive): \n");
        matprint(cols, rows, m_out_c);
	mvalidate(rows, cols, m_out_a, m_out_c, verbose);
        printf("Matrix after transposition by gpu (opt): \n");
        matprint(cols, rows, m_out_d);
	mvalidate(rows, cols, m_out_a, m_out_d, verbose);
    }

    // unallocate memory again
    free(m_in);
    free(m_out_a);
    free(m_out_c);
    free(m_out_d);

    return;
}




void task_two(int M, int N, bool verbose){
    printf("\n\nASSIGNMENT3 TASK2: MATRIX TRANSPOSITION AS PREPROCESSING\n");
    // Matrix Transposition during or as a pre-computatiion

    const int rows = M;
    const int cols = N;
    const char s_valid[10]   = "--  VALID";
    const char s_invalid[10] = "--INVALID";
    const int arr_size = rows * cols;
    float *m_in, *m_out_a, *m_out_c, *m_out_d;
    m_in    = (float*) malloc(arr_size * sizeof(float));
    m_out_a = (float*) malloc(arr_size * sizeof(float));
    m_out_c = (float*) malloc(arr_size * sizeof(float));
    m_out_d = (float*) malloc(arr_size * sizeof(float));
    unsigned long int elapsed_a, elapsed_c, elapsed_d;
    bool valid_c, valid_d; 

    // INITIALIZE MATRIX (ARRAY) TO WORK ON
    for (int i=0; i<(arr_size); i++){
        m_in[i] =  2.0; 
    }

    // TASK 2 A)
    { 
        elapsed_a = matrix_accfun_cpu( rows, cols, m_in, m_out_a);
    }

    // TASK 2 B) OMITTED

    // TASK 2 C)
    { 
	const unsigned char version = 1;
        elapsed_c = matrix_accfun_gpu<float>(rows, cols, m_in, m_out_c, version); 
	valid_c   = mvalidate<float>(rows, cols, m_out_a, m_out_c);
    }

    // TASK 2 D)
    { 
	const unsigned char version = 2;
        elapsed_d = matrix_accfun_gpu<float>(rows, cols, m_in, m_out_d, version);  
	valid_d   = mvalidate<float>(rows, cols, m_out_a, m_out_d);
    }

    // The modified program (CUDA transpositions included) has about two 
    // times the number of global memory accesses of the original program. 
    // Does it run faster or slower than the original, and by how much
    // (for a suitably large N)?
    
    // print statistics
    const double gpu_speedup = ((double) elapsed_c) / ((double) elapsed_d + 1);
    const double cpu_speedup1 = ((double) elapsed_a) / ((double) elapsed_c + 1);
    const double cpu_speedup2 = ((double) elapsed_a) / ((double) elapsed_d + 1);

    printf("\nMatrix accfun on size %d x %d. Running times:\n", rows, cols);
    printf("CPU:           %10lu microsecs. \n", elapsed_a);
    printf("GPU first:     %10lu microsecs. %s\n", elapsed_c, (valid_c ? s_valid: s_invalid));
    printf("GPU second:    %10lu microsecs. %s\n", elapsed_d, (valid_d ? s_valid: s_invalid));

    printf("\nThis is a speedup of %7.2f, for second compared to first on GPU.\n",gpu_speedup);
    printf("... and a speedup of %7.2f, for GPU first  compared to CPU.\n",cpu_speedup1);
    printf("... and a speedup of %7.2f, for GPU second compared to CPU.\n",cpu_speedup2);
    
    if (verbose) {
        printf("Matrix accumulation function on following array: \n");
        matprint( rows, cols, m_in );
        printf("Matrix accumulation function by cpu (naïve): \n");
        matprint( rows, cols, m_out_a);
        printf("Matrix accumulation function by gpu (first version): \n");
        matprint( rows, cols, m_out_c);
	mvalidate(rows, cols, m_out_a, m_out_c, verbose);
        printf("Matrix accumulation function by gpu (second version): \n");
        matprint( rows, cols, m_out_d);
	mvalidate(rows, cols, m_out_a, m_out_d, verbose);
    }

    free(m_in);
    free(m_out_a);
    free(m_out_c);
    free(m_out_d);

}



void task_three(int M, int U, int N, bool verbose){
    printf("\n\nASSIGNMENT3 TASK3: MATRIX MULTIPLICATION\n");

    const unsigned int a_rows = M;
    const unsigned int a_cols = U;
    const unsigned int b_rows = U;
    const unsigned int b_cols = N;
    const char s_valid[10]   = "--  VALID";
    const char s_invalid[10] = "--INVALID";
    const unsigned int a_size   = a_rows * a_cols;
    const unsigned int b_size   = b_rows * b_cols;
    const unsigned int res_size = a_rows * b_cols;
    float *m_in_a, *m_in_b, *m_out_a, *m_out_c, *m_out_d;
    m_in_a  = (float*) malloc(  a_size * sizeof(float));
    m_in_b  = (float*) malloc(  b_size * sizeof(float));
    m_out_a = (float*) malloc(res_size * sizeof(float));
    m_out_c = (float*) malloc(res_size * sizeof(float));
    m_out_d = (float*) malloc(res_size * sizeof(float));
    
    // INITIALIZE MATRICES (ARRAYS) TO WORK ON
    for (int i=0; i<(a_rows); i++){
        for (int j=0; j<(a_cols); j++){
            m_in_a[i*a_cols+j] = 0;
            if (abs(i-j)<2) {
                m_in_a[i*a_cols+j] = (i==j) ? 3.5 : -1.0 ;
            }
        }
    }

    for (int i=0; i<(b_rows); i++){
        for (int j=0; j<(b_cols); j++){
            m_in_b[i*b_cols+j] = 0;
            if (abs(i-j)<2) {
                m_in_b[i*b_cols+j] = (i==j) ? 2.5 : -1.0 ;
            }
        }
    }

    // initiate timing variable, keep results for validation
    unsigned long int elapsed_a, elapsed_c, elapsed_d;
    bool valid_c, valid_d; 
    
    // TASK 3 A)
    { 
        elapsed_a = matmult_cpu<float>(a_rows, a_cols, m_in_a, b_rows, b_cols, m_in_b, m_out_a);
    }
    
    // TASK 3 B) OMITTED

    // TASK 3 C)
    { 
	const unsigned char version = 1;
        elapsed_c = matmult_gpu<float>(a_rows, a_cols, m_in_a, b_rows, b_cols, m_in_b, m_out_c, version);
	valid_c   = mvalidate( a_rows, b_cols, m_out_a, m_out_c);
    }

    // TASK 3 D)
    { 
	const unsigned char version = 2;
        elapsed_d = matmult_gpu(a_rows, a_cols, m_in_a, b_rows, b_cols, m_in_b, m_out_d, version);
	valid_d   = mvalidate(a_rows, b_cols, m_out_a, m_out_d);
    }

    // Measure and compare the various running times. 
    // How many GFlops does the naïve and optimized CUDA versions achieve?
    const unsigned int flops = a_rows * (a_cols+b_rows) * b_cols;
    float gflops_a = (float) flops / (elapsed_a * 1000);
    float gflops_c = (float) flops / (elapsed_c * 1000);
    float gflops_d = (float) flops / (elapsed_d * 1000);

    // print statistics
    const double gpu_speedup = ((double) elapsed_c) / ((double) elapsed_d + 1);
    const double cpu_speedup1 = ((double) elapsed_a) / ((double) elapsed_c + 1);
    const double cpu_speedup2 = ((double) elapsed_a) / ((double) elapsed_d + 1);
    printf("\nMatrix Multiplication on (%dx%d) x (%d,%d). Timings:\n",a_rows, a_cols, b_rows, b_cols);
    printf("CPU:           %10lu microsecs. \n", elapsed_a);
    printf("GPU naïve:     %10lu microsecs. %s\n", elapsed_c, (valid_c ? s_valid: s_invalid));
    printf("GPU optimized: %10lu microsecs. %s\n", elapsed_d, (valid_d ? s_valid: s_invalid));

    printf("\nGiga FLoatingpointOPerations per second:\n");
    printf("CPU:           %10.3f Gflop/s.\n", gflops_a);
    printf("GPU naïve:     %10.3f Gflop/s.\n", gflops_c);
    printf("GPU optimized: %10.3f Gflop/s.\n", gflops_d);

    printf("\nThis is a speedup of %7.2f, for second compared to first on GPU.\n",gpu_speedup);
    printf("... and a speedup of %7.2f, for GPU naive compared to CPU.\n",cpu_speedup1);
    printf("... and a speedup of %7.2f, for GPU opt   compared to CPU.\n",cpu_speedup2);
    

    if (verbose) {
        printf("Matrix multiplication on following array: \n");
        matprint( a_rows, a_cols, m_in_a );
        matprint( b_rows, b_cols, m_in_b );
        printf("Matrix multiplication by cpu (naïve): \n");
        matprint( a_rows, b_cols, m_out_a);
        printf("Matrix multiplication by gpu (naïve): \n");
        matprint( a_rows, b_cols, m_out_c);
	mvalidate(a_rows, b_cols, m_out_a, m_out_c, verbose);
        printf("Matrix multiplication by gpu (optimized): \n");
        matprint( a_rows, b_cols, m_out_d);
	mvalidate(a_rows, b_cols, m_out_a, m_out_d, verbose);
    }

    free(m_in_a);
    free(m_in_b);
    free(m_out_a);
    free(m_out_c);
    free(m_out_d);
}





void warmup(const unsigned int size){
    // performing max segment sum calculation for GPU warmup purpose
    int *h_in  = (int*) malloc( size*size * sizeof(int));
    int *h_out = (int*) malloc( size*size * sizeof(int));

    for(unsigned int i=0; i<size; i++) h_in[i] = 1;

    transpose_gpu<int>( size, size, h_in, h_out );

    free(h_in);
    free(h_out);
    return;
}
